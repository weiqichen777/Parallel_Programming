#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float stepX, float stepY, float lowerX, float lowerY, int resX, int resY, int count, int *img_d, int pitch, int group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if(tid < resX*resY/group_size){

		int my_start = tid * group_size;
		int my_end = my_start + group_size;

		for(int j=my_start; j<my_end; j++){

			int thisX = j % resX;	//nth in that row
			int thisY = j / resX;	//which row
			float x = lowerX + thisX * stepX;
			float y = lowerY + thisY * stepY;
			float z_re = x, z_im = y;

			int i;
			for(i=0; i<count; i++){

				if(z_re * z_re + z_im * z_im > 4.f){
					break;
				}

				float new_re = z_re * z_re - z_im * z_im;
				float new_im = 2.f * z_re * z_im;
				z_re = x + new_re;
				z_im = y + new_im;
			}

			// int index = thisY * resX + thisX;
			//img_d[index] = i;

			int *rowHead;
			rowHead = (int *)((char *)img_d + thisY * pitch);
			rowHead[thisX] = i;
		}

	}
}

// Host front-end function that allocates the memory and launches the GPU kernel
// x1, y1, x0, y0, output, width, height, maxIterations
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX; //dx
    float stepY = (upperY - lowerY) / resY; //dy

    int group_size = 16;
	int block_size = 256;
	int num_blocks = (resX * resY) / (block_size * group_size) + 1;

	int *output;
	hipHostAlloc(&output, resX * resY * sizeof(int), hipHostMallocDefault);

	size_t pitch;
	int *img_d;
	hipMallocPitch(&img_d, &pitch, resX * sizeof(int), resY);

	hipMemcpy2D(img_d, pitch, output, resX * sizeof(int), resX * sizeof(int), resY, hipMemcpyHostToDevice);

	mandelKernel<<<num_blocks, block_size>>>(stepX, stepY, lowerX, lowerY, resX, resY, maxIterations, img_d, pitch, group_size);

	hipMemcpy2D(output, resX * sizeof(int), img_d, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

	for(int i=0; i<resX*resY; i++){
		img[i] = output[i];
	}

	hipHostFree(output);
	hipFree(img_d);
}
